#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <inttypes.h>

#include <Windows.h>
#include <gl\GL.h>

#include <cuda_gl_interop.h>
#include <>

#define TL_FUNC_PREFIX __device__
#define TL_NO_TEXTURE_CALLBACKS
#define TL_THUNDERLOOM_IMPLEMENTATION
extern "C"{
#include "thunderloom.h"
}

__global__
void trace_rays(float3 sun_dir, uint8_t *pixels, tlWeaveParameters *params)
{
	int w = blockDim.x*gridDim.x;
	int h = blockDim.y*gridDim.y;
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	int i = x + y * w;
	float fx = 2.f*(float)x/(float)w - 1.f;
	float fy = 2.f*(float)y/(float)h - 1.f;

	tlIntersectionData intersection_data;
	intersection_data.uv_x = fx;
	intersection_data.uv_y = fy;
	intersection_data.wi_x = sun_dir.x;
	intersection_data.wi_y = sun_dir.y;
	intersection_data.wi_z = sun_dir.z;

	intersection_data.wo_x = 0.f;
	intersection_data.wo_y = 0.f;
	intersection_data.wo_z = 1.f;
	intersection_data.context = 0;

	tlColor col = tl_shade(intersection_data, params);
	float mult = 5.f;
	col.r *= mult;
	col.g *= mult;
	col.b *= mult;
	
	pixels[i * 4 + 0] = (uint8_t)(saturate(col.r)*255.f);
	pixels[i * 4 + 1] = (uint8_t)(saturate(col.g)*255.f);
	pixels[i * 4 + 2] = (uint8_t)(saturate(col.b)*255.f);
	pixels[i * 4 + 3] = 255;
}

#define CHECK_CUDA_CALL(call) if((err= call) != hipSuccess){ printf("Error %d %s!\n   %s\n", err,\
	hipGetErrorName(err), hipGetErrorString(err)); abort();}

struct tlRaytracerParameters
{
	tlWeaveParameters *d_weave_params;
	tlYarnType *d_yarn_types;
	PatternEntry *d_pattern;
};

extern "C"
struct tlRaytracerParameters *tl_raytracer_load_parameters(tlWeaveParameters *params)
{
	enum hipError_t err;

	int num_yarn_types = params->num_yarn_types;
	tlYarnType *d_yarn_types = 0;
	CHECK_CUDA_CALL(hipMalloc(&d_yarn_types, num_yarn_types * sizeof(tlYarnType)));
	CHECK_CUDA_CALL(hipMemcpy(d_yarn_types, params->yarn_types, num_yarn_types * sizeof(tlYarnType), hipMemcpyHostToDevice));

	int res_pattern = params->pattern_width*params->pattern_height;
	PatternEntry *d_pattern;
	CHECK_CUDA_CALL(hipMalloc(&d_pattern, res_pattern * sizeof(PatternEntry)));
	CHECK_CUDA_CALL(hipMemcpy(d_pattern, params->pattern, res_pattern * sizeof(PatternEntry), hipMemcpyHostToDevice));

	tlWeaveParameters tmp_params = *params;
	tmp_params.yarn_types = d_yarn_types;
	tmp_params.pattern = d_pattern;
	tlWeaveParameters *d_params;
	CHECK_CUDA_CALL(hipMalloc(&d_params, sizeof(tlWeaveParameters)));
	CHECK_CUDA_CALL(hipMemcpy(d_params, &tmp_params, sizeof(tlWeaveParameters), hipMemcpyHostToDevice));
	return 0;
}


extern "C"
void tl_raytracer_render_to_memory(float sun_x, float sun_y, int res,
	tlWeaveParameters *params, uint8_t *pixels)
{
	enum hipError_t err;

	int num_yarn_types = params->num_yarn_types;
	tlYarnType *d_yarn_types = 0;
	CHECK_CUDA_CALL(hipMalloc(&d_yarn_types, num_yarn_types * sizeof(tlYarnType)));
	CHECK_CUDA_CALL(hipMemcpy(d_yarn_types, params->yarn_types, num_yarn_types * sizeof(tlYarnType), hipMemcpyHostToDevice));

	int res_pattern = params->pattern_width*params->pattern_height;
	PatternEntry *d_pattern;
	CHECK_CUDA_CALL(hipMalloc(&d_pattern, res_pattern * sizeof(PatternEntry)));
	CHECK_CUDA_CALL(hipMemcpy(d_pattern, params->pattern, res_pattern * sizeof(PatternEntry), hipMemcpyHostToDevice));

	tlWeaveParameters tmp_params = *params;
	tmp_params.yarn_types = d_yarn_types;
	tmp_params.pattern = d_pattern;
	tlWeaveParameters *d_params;
	CHECK_CUDA_CALL(hipMalloc(&d_params, sizeof(tlWeaveParameters)));
	CHECK_CUDA_CALL(hipMemcpy(d_params, &tmp_params, sizeof(tlWeaveParameters), hipMemcpyHostToDevice));

	dim3 dim_block(res, 1, 1);
	dim3 dim_grid(1, res, 1);

	uint8_t *d_pixels;
	CHECK_CUDA_CALL(hipMalloc(&d_pixels, res*res * 4 * sizeof(uint8_t)));

	float3 sun_dir;
	sun_dir.x = sun_x;
	sun_dir.y = sun_y;
	sun_dir.z = 1.f - sqrtf(sun_x*sun_x + sun_y*sun_y);
	trace_rays <<<dim_grid, dim_block >>> (sun_dir, d_pixels, d_params);

	CHECK_CUDA_CALL(hipDeviceSynchronize());

	CHECK_CUDA_CALL(hipMemcpy(pixels, d_pixels, res*res * 4 * sizeof(uint8_t), hipMemcpyDeviceToHost));

	CHECK_CUDA_CALL(hipFree(d_pixels));
}


extern "C"
void tl_raytracer_render_to_opengl_pbo(float sun_x, float sun_y, int res,
	tlWeaveParameters *params, unsigned int pbo_gl)
{
	enum hipError_t err;

	uint8_t *d_pixels = 0;
	size_t size = 0;
	hipGraphicsResource_t pbo_cuda;
	CHECK_CUDA_CALL(hipGraphicsGLRegisterBuffer(&pbo_cuda, pbo_gl, hipGraphicsRegisterFlagsWriteDiscard));
	CHECK_CUDA_CALL(hipGraphicsMapResources(1, &pbo_cuda, 0));
	CHECK_CUDA_CALL(hipGraphicsResourceGetMappedPointer((void**)&d_pixels, &size, pbo_cuda));

	int num_yarn_types = params->num_yarn_types;
	tlYarnType *d_yarn_types = 0;
	CHECK_CUDA_CALL(hipMalloc(&d_yarn_types, num_yarn_types * sizeof(tlYarnType)));
	CHECK_CUDA_CALL(hipMemcpy(d_yarn_types, params->yarn_types, num_yarn_types * sizeof(tlYarnType), hipMemcpyHostToDevice));

	int res_pattern = params->pattern_width*params->pattern_height;
	PatternEntry *d_pattern;
	CHECK_CUDA_CALL(hipMalloc(&d_pattern, res_pattern * sizeof(PatternEntry)));
	CHECK_CUDA_CALL(hipMemcpy(d_pattern, params->pattern, res_pattern * sizeof(PatternEntry), hipMemcpyHostToDevice));

	tlWeaveParameters tmp_params = *params;
	tmp_params.yarn_types = d_yarn_types;
	tmp_params.pattern = d_pattern;
	tlWeaveParameters *d_params;
	CHECK_CUDA_CALL(hipMalloc(&d_params, sizeof(tlWeaveParameters)));
	CHECK_CUDA_CALL(hipMemcpy(d_params, &tmp_params, sizeof(tlWeaveParameters), hipMemcpyHostToDevice));

	dim3 dim_block(res, 1, 1);
	dim3 dim_grid(1, res, 1);

	float3 sun_dir;
	sun_dir.x = sun_x;
	sun_dir.y = sun_y;
	sun_dir.z = 1.f - sqrtf(sun_x*sun_x + sun_y*sun_y);
	trace_rays <<<dim_grid, dim_block >>> (sun_dir, d_pixels, d_params);

	CHECK_CUDA_CALL(hipGraphicsUnmapResources(1, &pbo_cuda, 0));
	CHECK_CUDA_CALL(hipGraphicsUnregisterResource(pbo_cuda));

	CHECK_CUDA_CALL(hipFree(d_yarn_types));
	CHECK_CUDA_CALL(hipFree(d_pattern));
	CHECK_CUDA_CALL(hipFree(d_params));
}

